#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// Include SSE intrinsics
#if defined(_MSC_VER)
#include <intrin.h>
#elif defined(__GNUC__) && (defined(__x86_64__) || defined(__i386__))
#include <immintrin.h>
#include <x86intrin.h>
#endif

// Include OpenMP
//#include <omp.h>
extern "C" {
#include "layers.h"
}

#include "volume.h"

//extern double volume_get(volume_t *v, int x, int y, int d);
// Performs the forward pass for a convolutional layer by convolving each one
// of the filters with a particular input, and placing the result in the output
// array.
//
// One way to think about convolution in this case is that we have one of the
// layer's filters (a 3D array) that is superimposed on one of the layer's
// inputs (a second 3D array) that has been implicitly padded with zeros. Since
// convolution is a sum of products (described below), we don't actually have
// to add any zeros to the input volume since those terms will not contribute
// to the convolution. Instead, for each position in the filter, we just make
// sure that we are in bounds for the input volume.
//
// Essentially, the filter is "sliding" across the input, in both the x and y
// directions, where we increment our position in each direction by using the
// stride parameter.
//
// At each position, we compute the sum of the elementwise product of the filter
// and the part of the array it's covering. For instance, let's consider a 2D
// case, where the filter (on the left) is superimposed on some part of the
// input (on the right).
//
//   Filter             Input
//  -1  0  1           1  2  3
//  -1  0  1           4  5  6
//  -1  0  1           7  8  9
//
// Here, the sum of the elementwise product is:
//    Filter[0][0] * Input[0][0] + Filter[0][1] * Input[0][1] + ...
//    = -1 * 1 + 0 * 2 + ... + 0 * 8 + 1 * 9
//    = 6
//
// The 3D case is essentially the same, we just have to sum over the other
// dimension as well. Also, since volumes are internally represented as 1D
// arrays, we must use the volume_get and volume_set commands to access elements
// at a coordinate (x, y, d). Finally, we add the corresponding bias for the
// filter to the sum before putting it into the output volume.
unsigned long int N=4096*4096;


/*
   Here, your GPU kernel, modify the function header as well as kernel launch
   */
   __global__ void doGPU(conv_layer_t *layers,volume_t *in, volume_t *out) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
	int  x ,y,in_x , in_y;
        double sum = 0.0;
	x= col + layers->pad;
	y =row + layers->pad;
	out_x = blockIdx.x;
	out_y = blockIdx.y;
        volume_t *filter = layers->filters[blockId.x];
        for(int fy = 0; fy < filter->height; fy++){
                int in_y = y + fy;
                for(int fx = 0; fx < filter->width; fx++) {
                        int in_x = x + fx;
                        if(in_y >= 0 && in_y < in->height && in_x >=0 && in_x < in->width){
                                for(int fd = 0; fd < filter->depth; fd++) {
                                        sum += filter->weights[((filter->width * fy) + fx) * filter->depth + fd] * in->weights[((in->width * in_y) + in_x) * in->depth + fd];
                                }
                        }
                }
        }

        sum += layers->biases->weights[blockIdx.x];
        out->weights[((out->width * out_y) + out_x) * out->depth + f_index] = sum; 
extern "C" {


void conv_forward_cu(conv_layer_t *l, volume_t **inputs, volume_t **outputs, int start, int end) {
	dim3 dimGrid(16, 1);
	dim3 dimBlock(32,32);
	doGPU<<<gridsize, blocksize>>>(d_conv_layer);
}

}